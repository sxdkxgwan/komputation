
#include <hip/hip_runtime.h>
__device__ float backwardRelu (float forward, float chain)
{

    if(forward > 0.0) {

        return chain;

    }
    else {

        return 0.0;

    }

}

extern "C"
__global__ void backwardReluKernel (int length, float *forward, float *chain, float *destination)
{

    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index < length) {

        destination[index] = backwardRelu(forward[index], chain[index]);

    }

}